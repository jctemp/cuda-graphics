#include "hip/hip_runtime.h"
/*
 * kernel.cpp
 *
 *  Created on: 24.10.2010
 *      Author: ahlers
 */

#include "kernel.h"
#include <ctime>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

using namespace std;

struct ModelParams {
  GLsizei nParticles;
  float maxPosition;
  float mass;
  float g;
  float epsSqr;
};

const ModelParams params{2048, 10.0f, 1.0f, 0.001f, 0.0001f};

/*
 * global variables
 */

constexpr float EPSILON{1e-5};
constexpr float G{6.6743e-11};
const size_t nBlocks = 16;

/*
 * local function prototypes
 */

// simple kernel for testing
__global__ void setPositionsKernel(float4 *positions, float time,
                                   float maxPosition);

// kernel for N-body dynamics
__global__ void updatePositionsKernel(float4 *positions, float *velocityPtr,
                                      float time, float maxPosition);

// integrate ODE of a given particle
__device__ void stepIntegration(float4 *position, float4 *velocity,
                                float timeDiff);

// apply reflective boundary conditions to a given particle
__device__ void applyReflectiveBoundaryConditions(float4 *position,
                                                  float4 *velocity,
                                                  float maxPosition);

// apply reflective boundary conditions to a given particle
__device__ void applyPeriodicBoundaryCondition(float4 *position,
                                               float4 *velocity,
                                               float maxPosition);

// compute the body-body interactions
__device__ void computeBodyBodyInteractions(float3 *acceleration,
                                            float4 const *position,
                                            float4 const *positions,
                                            size_t nBodies);

// compute the gravitation
__device__ void addBodyBodyGravitation(float3 *acceleration,
                                       float4 const *position,
                                       float4 const *otherPosition);

__device__ void stepIntegrationLeapfrog(float4 *position, float4 *velocity,
                                        float3 *acceleration, float timeDiff);

/*
 * global function definitions
 */

void initCUDA() {
  // check for CUDA devices
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  checkCUDAError("initCUDA()");
  if (deviceCount == 0) {
    cerr << "Error: no CUDA device found" << endl;
    exit(1);
  }

  // use CUDA device 0
  // cudaGLSetGLDevice(0);
  checkCUDAError("initCUDA()");

  printCUDAVersion();
}

void checkCUDAError(const char *functionName) {
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    cerr << endl
         << "CUDA error in " << functionName << ": "
         << hipGetErrorString(error) << endl;
    exit(1);
  }
}

void printCUDAVersion() {
  int driverVersion = 0, runtimeVersion = 0;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  cout << "CUDA driver version: " << driverVersion / 1000 << "."
       << driverVersion % 100 << endl;
  cout << "CUDA runtime version: " << runtimeVersion / 1000 << "."
       << runtimeVersion % 100 << endl;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  cout << "CUDA device 0: " << deviceProp.name << endl;
  cout << "  Compute capability: " << deviceProp.major << "."
       << deviceProp.minor << endl;
  cout << "  Number of multiprocessors: " << deviceProp.multiProcessorCount
       << endl;
}

void launchCudaKernel(cudaGraphicsRes_pt &positionCudaVBO,
                      float *velocityCudaPtr, size_t nParticles,
                      float maxPosition) {
  static float timeOld = clock() / static_cast<float>(CLOCKS_PER_SEC);
  size_t nBytes;
  float4 *positions;
  hipGraphicsMapResources(1, &positionCudaVBO, 0);
  checkCUDAError("launchCudaKernel()");
  hipGraphicsResourceGetMappedPointer((void **)&positions, &nBytes,
                                       positionCudaVBO);
  checkCUDAError("launchCudaKernel()");

  float time = clock() / static_cast<float>(CLOCKS_PER_SEC);
  // setPositionsKernel<<<nBlocks, nParticles / nBlocks>>>(positions, time,
  // maxPosition);
  updatePositionsKernel<<<nBlocks, nParticles / nBlocks>>>(
      positions, velocityCudaPtr, time - timeOld, maxPosition);
  checkCUDAError("launchCudaKernel()");
  timeOld = time;
  hipGraphicsUnmapResources(1, &positionCudaVBO, 0);
  checkCUDAError("launchCudaKernel()");
}

/*
 * local function definitions
 */

// simple kernel for testing
__global__ void setPositionsKernel(float4 *positions, float time,
                                   float maxPosition) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int nParticles = gridDim.x * blockDim.x;

  positions[idx].x =
      maxPosition * 0.5f *
      (__cosf(3 * (3.14159f * idx / (float)nParticles + time)) + 1.0f);
  positions[idx].y =
      maxPosition * 0.5f *
      (__cosf(4 * (3.14159f * idx / (float)nParticles + time)) + 1.0f);
  positions[idx].z = maxPosition * idx / (float)nParticles;
}

// kernel for N-body dynamics
__global__ void updatePositionsKernel(float4 *positions, float *velocityPtr,
                                      float timeDiff, float maxPosition) {
  unsigned int idx{blockIdx.x * blockDim.x + threadIdx.x};
  unsigned int nParticles = gridDim.x * blockDim.x;
  float4 position = positions[idx];
  float4 *velocities{(float4 *)velocityPtr};
  float3 acceleration{0.0f};

  computeBodyBodyInteractions(&acceleration, &position, positions, nParticles);

  // stepIntegration(&(positions[idx]), &(velocities[idx]), timeDiff);
  stepIntegrationLeapfrog(&(positions[idx]), &(velocities[idx]), &acceleration,
                          timeDiff);

  // applyReflectiveBoundaryConditions(&(positions[idx]), &(velocities[idx]),
  // maxPosition);
  applyPeriodicBoundaryCondition(&(positions[idx]), &(velocities[idx]),
                                 maxPosition);
}

__device__ void stepIntegration(float4 *position, float4 *velocity,
                                float timeDiff) {
  position->x += timeDiff * velocity->x;
  position->y += timeDiff * velocity->y;
  position->z += timeDiff * velocity->z;
}

__device__ void stepIntegrationLeapfrog(float4 *position, float4 *velocity,
                                        float3 *acceleration, float timeDiff) {
  velocity->x += timeDiff * acceleration->x;
  velocity->y += timeDiff * acceleration->y;
  velocity->z += timeDiff * acceleration->z;
  position->x += timeDiff * velocity->x;
  position->y += timeDiff * velocity->y;
  position->z += timeDiff * velocity->z;
}

__device__ void applyReflectiveBoundaryConditions(float4 *position,
                                                  float4 *velocity,
                                                  float maxPosition) {
  if (position->x < 0.0f) {
    position->x *= -1;
    velocity->x *= -1;
  } else if (position->x > maxPosition) {
    position->x = maxPosition - (position->x - maxPosition);
    velocity->x *= -1;
  }
  if (position->y < 0.0f) {
    position->y *= -1;
    velocity->y *= -1;
  } else if (position->y > maxPosition) {
    position->y = maxPosition - (position->y - maxPosition);
    velocity->y *= -1;
  }
  if (position->z < 0.0f) {
    position->z *= -1;
    velocity->z *= -1;
  } else if (position->z > maxPosition) {
    position->z = maxPosition - (position->z - maxPosition);
    velocity->z *= -1;
  }
}

__device__ void applyPeriodicBoundaryCondition(float4 *position,
                                               float4 *velocity,
                                               float maxPosition) {
  position->x = fmodf(position->x + maxPosition, maxPosition);
  position->y = fmodf(position->y + maxPosition, maxPosition);
  position->z = fmodf(position->z + maxPosition, maxPosition);
}

__device__ void computeBodyBodyInteractions(float3 *acceleration,
                                            float4 const *position,
                                            float4 const *positions,
                                            size_t nBodies) {
  for (int i = 0; i < nBodies; i++) {
    float4 otherPosition = positions[i];
    addBodyBodyGravitation(acceleration, position, &otherPosition);
  }
}

__device__ void addBodyBodyGravitation(float3 *acceleration,
                                       float4 const *position,
                                       float4 const *otherPosition) {
  float3 direction = {
      otherPosition->x - position->x,
      otherPosition->y - position->y,
      otherPosition->z - position->z,
  };
  float distSqrWEps = direction.x * direction.x + direction.y * direction.y +
                      direction.z * direction.z + EPSILON * EPSILON;
  float factor =
      G * otherPosition->w / sqrtf(distSqrWEps * distSqrWEps * distSqrWEps);
  acceleration->x += factor * direction.x;
  acceleration->y += factor * direction.y;
  acceleration->z += factor * direction.z;
}
